
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
<<<<<<< HEAD
#include <vector>

#include "caffe/layers/cudnn_sigmoid_layer.hpp"
=======
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9

namespace caffe {

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
<<<<<<< HEAD
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
=======
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        CUDNN_ACTIVATION_SIGMOID,
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
<<<<<<< HEAD
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
=======
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
}

template <typename Dtype>
void CuDNNSigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
<<<<<<< HEAD
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
=======
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        CUDNN_ACTIVATION_SIGMOID,
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
<<<<<<< HEAD
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
=======
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSigmoidLayer);

}  // namespace caffe
#endif
