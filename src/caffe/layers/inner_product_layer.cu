#include <vector>

<<<<<<< HEAD
#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"
=======
#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9

namespace caffe {

template <typename Dtype>
void InnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
<<<<<<< HEAD
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
=======
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
      bottom_data, weight, (Dtype)0., top_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
  }
}

template <typename Dtype>
void InnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
<<<<<<< HEAD
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          K_, N_, M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
=======
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff, bottom_data, (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
<<<<<<< HEAD
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          M_, K_, N_,
          (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          M_, K_, N_,
         (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
=======
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff, this->blobs_[0]->gpu_data(), (Dtype)0.,
        bottom[0]->mutable_gpu_diff());
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductLayer);

}  // namespace caffe
