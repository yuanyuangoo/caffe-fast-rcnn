
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

<<<<<<< HEAD
#include "caffe/layers/cudnn_conv_layer.hpp"
=======
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
<<<<<<< HEAD
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
=======
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    size_t workspace_limit_bytes = this->kernel_h_ *
                                   this->kernel_w_ *
                                   this->channels_ *
                                   sizeof(int) + 1;

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      cudnnConvolutionFwdAlgo_t algo;

      // pick the convolution algorithm
      // TODO(shelhamer) this should be done during reshape
      // TODO(shelhamer) the choice of automatic or manual algorithm picking
      // should be exposed in proto
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_limit_bytes,  // memoryLimitInBytes,
        &algo));

      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes_temp = 0;

      CUDNN_CHECK(cudnnGetConvolutionForwardWorkspaceSize(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        algo,
        &workspaceSizeInBytes_temp));

      if (workspaceSizeInBytes_temp > workspaceSizeInBytes) {
        workspaceSizeInBytes = workspaceSizeInBytes_temp;
        // free the existing workspace and allocate a new (larger) one
        cudaFree(this->workspace);
        cudaError_t err = cudaMalloc(&(this->workspace), workspaceSizeInBytes);
        if (err != cudaSuccess) {
          // force zero memory path
          algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
          workspace = NULL;
          workspaceSizeInBytes = 0;
        }
      }

>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
<<<<<<< HEAD
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
=======
            filter_desc_, weight + weight_offset_ * g,
            conv_descs_[i],
            algo, workspace, workspaceSizeInBytes,
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
<<<<<<< HEAD
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
=======
        CUDNN_CHECK(cudnnAddTensor(handle_[g], CUDNN_ADD_SAME_C,
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
<<<<<<< HEAD
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
=======
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ + g],
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
<<<<<<< HEAD
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
=======
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + weight_offset_ * g));
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
<<<<<<< HEAD
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
=======
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
>>>>>>> 28a579eaf0668850705598b3075b8969f22226d9
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
